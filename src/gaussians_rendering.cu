#include "hip/hip_runtime.h"

// CHECKLICENSE
// much stuff from https://github.com/mkkellogg/GaussianSplats3D/blob/main/LICENSE
// (MIT license)

#include <cooperative_groups/scan.h>
#include <cooperative_groups/reduce.h>

#define CUB_DISABLE_BF16_SUPPORT

#define GLM_FORCE_CUDA
#define CUDA_VERSION 12000

namespace std{
	using size_t = ::size_t;
};

using namespace std;

#include <hip/hip_fp16.h>

#include "./libs/glm/glm/glm.hpp"
#include "./libs/glm/glm/gtc/matrix_transform.hpp"
// #include "./libs/glm/glm/gtc/matrix_access.hpp"
// #include "./libs/glm/glm/gtx/transform.hpp"
#include "./libs/glm/glm/gtc/quaternion.hpp"
#include "./libs/glm/glm/gtx/matrix_decompose.hpp"

#include "utils.cuh"
#include "HostDeviceInterface.h"
#include "math.cuh"

namespace cg = cooperative_groups;

using glm::ivec2;
using glm::vec2;
using glm::vec3;
using glm::vec4;
using glm::mat3;
using glm::mat4;
using glm::quat;
using glm::dot;
using glm::transpose;
using glm::inverse;


constexpr int VIEWMODE_DESKTOP = 0;
constexpr int VIEWMODE_DESKTOP_VR = 1;
constexpr int VIEWMODE_IMMERSIVE_VR = 2;
// constexpr uint32_t BACKGROUND_COLOR = 0xff000000;
// constexpr uint32_t BACKGROUND_COLOR = 0xffffffff;
constexpr uint32_t BACKGROUND_COLOR = 0xff443322;
constexpr uint64_t DEFAULT_PIXEL = (uint64_t(Infinity) << 32) | BACKGROUND_COLOR;

constexpr float3 SPECTRAL[11] = {
	float3{158,1,66},
	float3{213,62,79},
	float3{244,109,67},
	float3{253,174,97},
	float3{254,224,139},
	float3{255,255,191},
	float3{230,245,152},
	float3{171,221,164},
	float3{102,194,165},
	float3{50,136,189},
	float3{94,79,162},
};

__device__ uint64_t g_uclosest;

__device__ double g_x;
__device__ double g_y;
__device__ double g_z;
__device__ uint32_t g_numSelected;
__device__ bool dbg_bool;
__device__ uint32_t g_counter;

constexpr int dbgtile_x = 60;
constexpr int dbgtile_y = 50;
constexpr float basisvector_encoding_factor = 20.0f;

glm::i16vec2 encode_basisvector_i16vec2(vec2 basisvector){

	float length = glm::length(basisvector);
	float angle = atan2(basisvector.y, basisvector.x);

	int16_t ilength = clamp(length * basisvector_encoding_factor, 0.0f, 60'000.0f);
	int16_t iangle = angle * 10'000.0f;

	return {iangle, ilength};
}

vec2 decode_basisvector_i16vec2(glm::i16vec2 encoded){

	float length = float(encoded.y) / basisvector_encoding_factor;
	float angle = float(encoded.x) / 10'000.0f;

	float x = cos(angle);
	float y = sin(angle);

	return vec2{x, y} * length;
}

__half2 decode_basisvector_i16vec2_half2(glm::i16vec2 encoded){

	float length = float(encoded.y) / basisvector_encoding_factor;
	float angle = float(encoded.x) / 10'000.0f;

	float x = cos(angle) * length;
	float y = sin(angle) * length;

	return __half2{x, y};
}

#if defined(STAGEDATA_16BYTE)

	void encode_stagedata(StageData& stagedata, vec2 a, vec2 b, float depth){
		stagedata.basisvector1_encoded = encode_basisvector_i16vec2(a);
		// stagedata.basisvector2_encoded = encode_basisvector_i16vec2(b);

		vec2 b_encoded = encode_basisvector_i16vec2(b);
		stagedata.basisvector2_encoded = b_encoded.y;
		
		// stagedata.depth = depth;

		__half hdepth = depth;
		memcpy(&stagedata.depth_encoded, &hdepth, 2);
	}

	void decode_stagedata(StageData stagedata, vec2& a, vec2& b, float& depth){
		a = decode_basisvector_i16vec2(stagedata.basisvector1_encoded);
		// b = decode_basisvector_i16vec2(stagedata.basisvector2_encoded);

		float b_length = float(stagedata.basisvector2_encoded) / basisvector_encoding_factor;
		b = normalize(vec2{-a.y, a.x}) * b_length;

		// depth = stagedata.depth;

		__half hdepth;
		memcpy(&hdepth, &stagedata.depth_encoded, 2);
		depth = hdepth;
	}

#elif defined(STAGEDATA_20BYTE)

	void encode_stagedata(StageData& stagedata, vec2 a, vec2 b, float depth){
		stagedata.basisvector1_encoded = encode_basisvector_i16vec2(a);
		stagedata.basisvector2_encoded = encode_basisvector_i16vec2(b);

		stagedata.depth = depth;
	}

	void decode_stagedata(StageData stagedata, vec2& a, vec2& b, float& depth){
		a = decode_basisvector_i16vec2(stagedata.basisvector1_encoded);
		b = decode_basisvector_i16vec2(stagedata.basisvector2_encoded);

		depth = stagedata.depth;
	}

#elif defined(STAGEDATA_24BYTE)

	void encode_stagedata(StageData& stagedata, vec2 a, vec2 b, float depth){
		stagedata.basisvector1_encoded = encode_basisvector_i16vec2(a);
		stagedata.basisvector2_encoded = encode_basisvector_i16vec2(b);

		stagedata.depth = depth;
	}

	void decode_stagedata(StageData stagedata, vec2& a, vec2& b, float& depth){
		a = decode_basisvector_i16vec2(stagedata.basisvector1_encoded);
		b = decode_basisvector_i16vec2(stagedata.basisvector2_encoded);

		depth = stagedata.depth;
	}

#endif

// based on https://github.com/sjtuzq/point-radiance/blob/main/modules/sh.py (Differentiable Point-Based Radiance Fields for Efficient View Synthesis)
// and Inria 3DGS forward.cu - computeColorFromSH() (3D Gaussian Splatting for Real-Time Radiance Field Rendering)
vec3 getHarmonics(
	int degree, 
	int numCoefficients, 
	vec3 camdir,
	vec3* sh
){
	// SH coefficients from https://github.com/sjtuzq/point-radiance/blob/main/modules/sh.py
	// (BSD 2-clause license), Copyright 2021 The PlenOctree Authors.
	// constexpr float C0 = 0.28209479177387814;
	constexpr float C1 = 0.4886025119029199;
	constexpr float C2[] = {
		 1.0925484305920792f,
		-1.0925484305920792f,
		 0.31539156525252005f,
		-1.0925484305920792f,
		 0.5462742152960396f
	};
	constexpr float C3[] = {
		-0.5900435899266435f,
		 2.890611442640554f,
		-0.4570457994644658f,
		 0.3731763325901154f,
		-0.4570457994644658f,
		 1.445305721320277f,
		-0.5900435899266435f
	};
	constexpr float C4[] = {
		 2.5033429417967046f,
		-1.7701307697799304f,
		 0.9461746957575601f,
		-0.6690465435572892f,
		 0.10578554691520431f,
		-0.6690465435572892f,
		 0.47308734787878004f,
		-1.7701307697799304f,
		 0.6258357354491761f,
	};

	float xx = camdir.x * camdir.x;
	float yy = camdir.y * camdir.y;
	float zz = camdir.z * camdir.z;
	float xy = camdir.x * camdir.y; 
	float yz = camdir.y * camdir.z; 
	float xz = camdir.x * camdir.z;

	vec3 result = {0.0f, 0.0f, 0.0f};

	if(degree > 0){
		result = result -
			C1 * camdir.y * sh[0] + 
			C1 * camdir.z * sh[1] - 
			C1 * camdir.x * sh[2];
	}

	if(degree > 1){
		result = result +
			C2[0] * xy * sh[3] +
			C2[1] * yz * sh[4] +
			C2[2] * (2.0f * zz - xx - yy) * sh[5] +
			C2[3] * xz * sh[6] +
			C2[4] * (xx - yy) * sh[7];
	}

	if(degree > 2){
		result = result +
			C3[0] * camdir.y * (3.0f * xx - yy) * sh[8] +
			C3[1] * xy * camdir.z * sh[9] +
			C3[2] * camdir.y * (4.0f * zz - xx - yy) * sh[10] +
			C3[3] * camdir.z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[11] +
			C3[4] * camdir.x * (4.0f * zz - xx - yy) * sh[12] +
			C3[5] * camdir.z * (xx - yy) * sh[13] +
			C3[6] * camdir.x * (xx - 3.0f * yy) * sh[14];
	}

	return result;
}



// We need the same iteration logic for "touched" tiles in multiple kernels. 
template <typename Function>
void forEachTouchedTile(vec2 splatCoord, vec2 basisVector1, vec2 basisVector2, RenderTarget target, Function f){

	// use approximate tile-splat intersection
	float quadHalfWidth  = sqrt(basisVector1.x * basisVector1.x + basisVector2.x * basisVector2.x);
	float quadHalfHeight = sqrt(basisVector1.y * basisVector1.y + basisVector2.y * basisVector2.y);

	ivec2 tile_start = {
		(splatCoord.x - quadHalfWidth) / TILE_SIZE_3DGS,
		(splatCoord.y - quadHalfHeight) / TILE_SIZE_3DGS};
	ivec2 tile_end = {
		(splatCoord.x + quadHalfWidth) / TILE_SIZE_3DGS,
		(splatCoord.y + quadHalfHeight) / TILE_SIZE_3DGS};

	float tiles_x = ceil(target.width / float(TILE_SIZE_3DGS));
	float tiles_y = ceil(target.height / float(TILE_SIZE_3DGS));

	if(tile_end.x < 0 || tile_start.x >= tiles_x) return;
	if(tile_end.y < 0 || tile_start.y >= tiles_y) return;

	tile_start.x = max(tile_start.x, 0);
	tile_end.x = min(tile_end.x, int(tiles_x) - 1);

	tile_start.y = max(tile_start.y, 0);
	tile_end.y = min(tile_end.y, int(tiles_y) - 1);

	// float diag = 22.627416997969522f;  // sqrt(16.0f * 16.0f + 16.0f * 16.0f);
	float tileRadius = 11.313708498984761f + 0.0f; // sqrt(8.0f * 8.0f + 8.0f * 8.0f);
	for(int tile_x = tile_start.x; tile_x <= tile_end.x; tile_x++)
	for(int tile_y = tile_start.y; tile_y <= tile_end.y; tile_y++)
	{
		vec2 tilePos = {tile_x * 16.0f + 8.0f, tile_y * 16.0f + 8.0f}; 

		bool intersectsTile = intersection_circle_splat(
			tilePos, tileRadius, 
			splatCoord, 
			basisVector1, 
			basisVector2
		);

		// intersectsTile = true;

		if(intersectsTile){
			f(tile_x, tile_y);
		}
	}
}

template <typename Function>
void forEachTouchedTile(int mode, vec2 splatCoord, vec2 basisVector1, vec2 basisVector2, RenderTarget target, Function f){

	if(mode == INTERSECTION_APPROXIMATE){
		forEachTouchedTile_approx(splatCoord, basisVector1, basisVector2, target, f);
	}else if(mode == INTERSECTION_3DGS){
		forEachTouchedTile_approx(splatCoord, basisVector1, basisVector2, target, f);
	}else if(mode == INTERSECTION_TIGHTBB){
		forEachTouchedTile_approx(splatCoord, basisVector1, basisVector2, target, f);
	}

}

// We need the same iteration logic for "touched" tiles in multiple kernels. 
template <typename Function>
void forEachTouchedTile_approx(vec2 splatCoord, vec2 basisVector1, vec2 basisVector2, RenderTarget target, Function f){

	// use approximate tile-splat intersection
	float quadHalfWidth  = sqrt(basisVector1.x * basisVector1.x + basisVector2.x * basisVector2.x);
	float quadHalfHeight = sqrt(basisVector1.y * basisVector1.y + basisVector2.y * basisVector2.y);

	ivec2 tile_start = {
		(splatCoord.x - quadHalfWidth) / TILE_SIZE_3DGS,
		(splatCoord.y - quadHalfHeight) / TILE_SIZE_3DGS};
	ivec2 tile_end = {
		(splatCoord.x + quadHalfWidth) / TILE_SIZE_3DGS,
		(splatCoord.y + quadHalfHeight) / TILE_SIZE_3DGS};

	float tiles_x = ceil(target.width / float(TILE_SIZE_3DGS));
	float tiles_y = ceil(target.height / float(TILE_SIZE_3DGS));

	if(tile_end.x < 0 || tile_start.x >= tiles_x) return;
	if(tile_end.y < 0 || tile_start.y >= tiles_y) return;

	tile_start.x = max(tile_start.x, 0);
	tile_end.x = min(tile_end.x, int(tiles_x) - 1);

	tile_start.y = max(tile_start.y, 0);
	tile_end.y = min(tile_end.y, int(tiles_y) - 1);

	// float diag = 22.627416997969522f;  // sqrt(16.0f * 16.0f + 16.0f * 16.0f);
	float tileRadius = 11.313708498984761f + 0.0f; // sqrt(8.0f * 8.0f + 8.0f * 8.0f);
	for(int tile_x = tile_start.x; tile_x <= tile_end.x; tile_x++)
	for(int tile_y = tile_start.y; tile_y <= tile_end.y; tile_y++)
	{
		vec2 tilePos = {tile_x * 16.0f + 8.0f, tile_y * 16.0f + 8.0f}; 

		bool intersectsTile = intersection_circle_splat(
			tilePos, tileRadius, 
			splatCoord, 
			basisVector1, 
			basisVector2
		);

		// intersectsTile = true;

		if(intersectsTile){
			f(tile_x, tile_y);
		}
	}
}

// We need the same iteration logic for "touched" tiles in multiple kernels. 
template <typename Function>
void forEachTouchedTile_3dgs(vec2 splatCoord, vec2 basisVector1, vec2 basisVector2, RenderTarget target, Function f){

	// Use 3DGS-style bounding box around ellipses' bounding sphere
	float maxRadius = max(length(basisVector1), length(basisVector2));

	ivec2 tile_start = {
		(splatCoord.x - maxRadius) / TILE_SIZE_3DGS,
		(splatCoord.y - maxRadius) / TILE_SIZE_3DGS,
	};
	ivec2 tile_end = {
		(splatCoord.x + maxRadius) / TILE_SIZE_3DGS,
		(splatCoord.y + maxRadius) / TILE_SIZE_3DGS,
	};

	float tiles_x = ceil(target.width / float(TILE_SIZE_3DGS));
	float tiles_y = ceil(target.height / float(TILE_SIZE_3DGS));

	if(tile_end.x < 0 || tile_start.x >= tiles_x) return;
	if(tile_end.y < 0 || tile_start.y >= tiles_y) return;

	tile_start.x = max(tile_start.x, 0);
	tile_end.x = min(tile_end.x, int(tiles_x) - 1);

	tile_start.y = max(tile_start.y, 0);
	tile_end.y = min(tile_end.y, int(tiles_y) - 1);

	for(int tile_x = tile_start.x; tile_x <= tile_end.x; tile_x++)
	for(int tile_y = tile_start.y; tile_y <= tile_end.y; tile_y++)
	{
		f(tile_x, tile_y);
	}
}

template <typename Function>
void forEachTouchedTile_tightBB(vec2 splatCoord, vec2 basisVector1, vec2 basisVector2, RenderTarget target, Function f){

	// Use tight bounding box around ellipse
	float quadHalfWidth  = sqrt(basisVector1.x * basisVector1.x + basisVector2.x * basisVector2.x);
	float quadHalfHeight = sqrt(basisVector1.y * basisVector1.y + basisVector2.y * basisVector2.y);

	ivec2 tile_start = {
		(splatCoord.x - quadHalfWidth) / TILE_SIZE_3DGS,
		(splatCoord.y - quadHalfHeight) / TILE_SIZE_3DGS};
	ivec2 tile_end = {
		(splatCoord.x + quadHalfWidth) / TILE_SIZE_3DGS,
		(splatCoord.y + quadHalfHeight) / TILE_SIZE_3DGS};

	float tiles_x = ceil(target.width / float(TILE_SIZE_3DGS));
	float tiles_y = ceil(target.height / float(TILE_SIZE_3DGS));

	if(tile_end.x < 0 || tile_start.x >= tiles_x) return;
	if(tile_end.y < 0 || tile_start.y >= tiles_y) return;

	tile_start.x = max(tile_start.x, 0);
	tile_end.x = min(tile_end.x, int(tiles_x) - 1);

	tile_start.y = max(tile_start.y, 0);
	tile_end.y = min(tile_end.y, int(tiles_y) - 1);

	for(int tile_x = tile_start.x; tile_x <= tile_end.x; tile_x++)
	for(int tile_y = tile_start.y; tile_y <= tile_end.y; tile_y++)
	{
		f(tile_x, tile_y);
	}
}



bool isDebugTile(int x, int y){
	return x == dbgtile_x && y == dbgtile_y;
}

// stages the model for rasterization, meaning it creates tile fragments for each tile the splat overlaps, and adds it to the staging buffer.
// Much of the math here is from https://github.com/mkkellogg/GaussianSplats3D
extern "C" __global__
void kernel_stageSplats(
	CommonLaunchArgs args,
	RenderTarget target,
	ColorCorrection colorCorrection,
	GaussianData model,
	// out
	uint32_t* visibleSplatCounter,
	uint32_t* numTilefragments,
	uint32_t* numTilefragments_splatwise,
	float* staging_depth,
	StageData* staging_data,
	uint32_t* ordering
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();
	int splatIndex = grid.thread_rank();

	if(splatIndex >= model.count) return;

	mat4 world = model.transform;
	mat4 view = target.view;
	mat4 proj = target.proj;
	mat4 worldView = view * world;
	mat4 worldViewProj = proj * view * world;

	vec3 splatPos = model.position[splatIndex];
	vec4 worldPos = world * vec4(splatPos, 1.0f);
	vec4 viewPos = view * worldPos;
	vec4 ndc = proj * viewPos;

	ndc.x = ndc.x / ndc.w;
	ndc.y = ndc.y / ndc.w;
	ndc.z = ndc.z / ndc.w;


	// if(grid.thread_rank() == 0) printf("=============\n");
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.position));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.scale));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.quaternion));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.color));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.sphericalHarmonics));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.cov3d));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.depth));
	// if(grid.thread_rank() == 0) printf("%llu\n", uint64_t(model.flags));

	if(ndc.w <= 0.0f) return;
	// Early-Discard gaussians whose position is far outside the frustum 
	if(ndc.x < -1.5f || ndc.x >  1.5f) return;
	if(ndc.y < -1.5f || ndc.y >  1.5f) return;

	vec4 color = model.color[splatIndex].normalized();

	// WIP: SHs not yet robust under model and splat transformations
	constexpr int updateOverXFrames = 10;
	if(splatIndex % updateOverXFrames == args.uniforms.frameCount % updateOverXFrames)
	// if(false)
	if(model.shDegree > 0 ){
		// if(splatIndex == 0) printf("model.numSHCoefficients: %d \n", model.numSHCoefficients);
		// int64_t offset = splatIndex * 45;
		int64_t offset = splatIndex * model.numSHCoefficients;
		vec3* shs = (vec3*)(model.sphericalHarmonics + offset);
		vec3 camPos = inverse(target.view) * vec4(0.0f, 0.0f, 0.0f, 1.0f);
		

		vec4 harmonics = vec4{0.0f};
		// camdir = normalize(vec3{
		// 	cos(0.5f * floor(args.uniforms.time)), 
		// 	sin(0.5f * floor(args.uniforms.time)), 
		// 	 0.0f
		// });

		vec3 camdir = normalize(vec3(worldPos) - camPos);
		camdir = vec3((inverse(model.transform)) * vec4(camdir, 0.0f));
		camdir = normalize(camdir);
		harmonics = vec4(getHarmonics(model.shDegree, model.numSHCoefficients, camdir, shs), 0.0f);

		// vec3 d_scale;
		// quat d_rotation;
		// vec3 d_translation;
		// vec3 d_skew;
		// vec4 d_perspective;
		// glm::decompose(model.transform, d_scale, d_rotation, d_translation, d_skew, d_perspective);
		// d_rotation = glm::conjugate(d_rotation);

		// mat3 rotation = glm::mat3_cast(d_rotation);

		// vec3 shs_transformed[15];
		// rotateSH(model.shDegree, shs, shs_transformed, rotation);

		// harmonics = vec4(getHarmonics(model.shDegree, model.numSHCoefficients, camdir, shs_transformed), 0.0f);

		vec4 colBaked = color + 1.0f * harmonics;
		// vec4 colBaked = 50.0f * harmonics;
		colBaked = glm::max(colBaked, 0.0f);
		colBaked.a = color.a;

		uint32_t C = 
			(uint32_t(clamp(256.0f * colBaked.r, 0.0f, 255.0f)) <<  0) |
			(uint32_t(clamp(256.0f * colBaked.g, 0.0f, 255.0f)) <<  8) |
			(uint32_t(clamp(256.0f * colBaked.b, 0.0f, 255.0f)) << 16) |
			(uint32_t(clamp(256.0f * colBaked.a, 0.0f, 255.0f)) << 24);
		model.color_resolved[splatIndex] = C;

		// color = colBaked;
	}

	if(model.shDegree > 0 )
	{
		uint32_t C = model.color_resolved[splatIndex];

		color.r = float((C >>  0) & 0xff) / 256.0f;
		color.g = float((C >>  8) & 0xff) / 256.0f;
		color.b = float((C >> 16) & 0xff) / 256.0f;
		color.a = float((C >> 24) & 0xff) / 256.0f;
	}


	vec4 quat = model.quaternion[splatIndex];

	// int8_t qw = quat.w * 127.0f;

	// // __half qx = __half(quat.x);
	// // __half qy = __half(quat.y);
	// // __half qz = __half(quat.z);
	// // __half qw = __half(quat.w);

	auto q = glm::quat(quat.x, quat.y, quat.z, quat.w);
	mat3 rotation = glm::mat3_cast(q);

	mat3 scale = mat3(1.0f);
	scale[0][0] = model.scale[splatIndex].x;
	scale[1][1] = model.scale[splatIndex].y;
	scale[2][2] = model.scale[splatIndex].z;

	mat3 cov3D = rotation * scale * scale * transpose(rotation);

	// check lower bitrates
	// float* floats = (float*)&cov3D;
	// for(int i = 0; i < 9; i++){
	// 	floats[i] = __half(floats[i]);
	// 	// floats[i] = clamp(floats[i], -10.0f, 10.0f);
	// }

	// model.cov3d[index].m11 = cov3D[0][0];
	// model.cov3d[index].m12 = cov3D[0][1];
	// model.cov3d[index].m13 = cov3D[0][2];
	// model.cov3d[index].m22 = cov3D[1][1];
	// model.cov3d[index].m23 = cov3D[1][2];
	// model.cov3d[index].m33 = cov3D[2][2];

	// auto cov3Del = model.cov3d[index];
	// mat3 cov3D;
	// cov3D[0][0] = cov3Del.m11;
	// cov3D[0][1] = cov3Del.m12;
	// cov3D[0][2] = cov3Del.m13;

	// cov3D[1][0] = cov3Del.m12;
	// cov3D[1][1] = cov3Del.m22;
	// cov3D[1][2] = cov3Del.m23;

	// cov3D[2][0] = cov3Del.m13;
	// cov3D[2][1] = cov3Del.m23;
	// cov3D[2][2] = cov3Del.m33;

	vec2 focal = vec2(
		proj[0][0] * target.width * 0.5f,
		proj[1][1] * target.height * 0.5f
	);
	float s = 1.0f / (viewPos.z * viewPos.z);
	mat3 J = mat3(
		focal.x / viewPos.z   , 0.0f                  , -(focal.x * viewPos.x) * s,
		0.0f                  , focal.y / viewPos.z   , -(focal.y * viewPos.y) * s,
		0.0f                  , 0.0f                  , 0.0f
	);

	mat3 W = transpose(mat3(worldView));
	mat3 T = W * J;

	mat3 cov2Dm = transpose(T) * cov3D * T;
	cov2Dm[0][0] += 0.2f;
	cov2Dm[1][1] += 0.2f;

	vec3 cov2Dv = vec3(cov2Dm[0][0], cov2Dm[0][1], cov2Dm[1][1]);

	float a = cov2Dv.x;
	float d = cov2Dv.z;
	float b = cov2Dv.y;
	float D = a * d - b * b;
	float trace = a + d;
	float traceOver2 = 0.5f * trace;
	float term2 = sqrt(max(0.1f, traceOver2 * traceOver2 - D));
	float eigenValue1 = traceOver2 + term2;
	float eigenValue2 = traceOver2 - term2;

	// eigenValue1 = max(eigenValue1, 0.1f);
	// eigenValue2 = max(eigenValue2, 0.1f);

	if(args.uniforms.cullSmallSplats){
		// clip tiny gaussians
		if(eigenValue1 < 0.02f) return;
		if(eigenValue2 < 0.02f) return;
	}else{
		eigenValue1 = max(eigenValue1, 0.001f);
		eigenValue2 = max(eigenValue2, 0.001f);

		// reduce opacity of small splats
		if(eigenValue1 == 0.001f) color.a *= 0.15f;
		if(eigenValue2 == 0.001f) color.a *= 0.15f;

		// cull small splats with little opacity?
	}
	if(color.a < 10.0f / 256.0f) return;

	if(args.uniforms.makePoints){
		if(eigenValue1 < 0.105f) return;
		if(eigenValue2 < 0.105f) return;
	}

	vec2 eigenVector1 = normalize(vec2(b, eigenValue1 - a));
	vec2 eigenVector2 = vec2(eigenVector1.y, -eigenVector1.x);

	float splatScale = args.uniforms.splatSize;
	const float sqrt8 = sqrt(8.0f);

	vec2 _basisVector1 = eigenVector1 * splatScale * min(sqrt8 * sqrt(eigenValue1), MAX_SCREENSPACE_SPLATSIZE);
	vec2 _basisVector2 = eigenVector2 * splatScale * min(sqrt8 * sqrt(eigenValue2), MAX_SCREENSPACE_SPLATSIZE);

	// We are using quantized basisvectors elsewhere, 
	// so we must make sure to also use them in this kernel to obtain matching results
	vec2 basisVector1, basisVector2;
	StageData tmp;
	float tmpdepth;
	encode_stagedata(tmp, _basisVector1, _basisVector2, tmpdepth);
	decode_stagedata(tmp, basisVector1, basisVector2, tmpdepth);

	// discard small splats with low opacity
	if(args.uniforms.cullSmallSplats)
	if(length(_basisVector1) < 5.0f)
	if(length(_basisVector2) < 5.0f)
	{
		if(color.a < 0.2f) return;
	}

	float depth = ndc.w;
	vec2 _pixelCoord = {
		((ndc.x) * 0.5f + 0.5f) * target.width,
		((ndc.y) * 0.5f + 0.5f) * target.height
	};

	glm::i16vec2 pixelCoord_encoded = _pixelCoord * 10.0f;
	vec2 pixelCoord = vec2(pixelCoord_encoded) / 10.0f;

	float quadHalfWidth  = sqrt(basisVector1.x * basisVector1.x + basisVector2.x * basisVector2.x);
	float quadHalfHeight = sqrt(basisVector1.y * basisVector1.y + basisVector2.y * basisVector2.y);

	ivec2 tile_start = {
		(pixelCoord.x - quadHalfWidth) / TILE_SIZE_3DGS,
		(pixelCoord.y - quadHalfHeight) / TILE_SIZE_3DGS};
	ivec2 tile_end = {
		(pixelCoord.x + quadHalfWidth) / TILE_SIZE_3DGS,
		(pixelCoord.y + quadHalfHeight) / TILE_SIZE_3DGS};
	ivec2 tile_size = (tile_end - tile_start) + 1;

	float tiles_x = ceil(target.width / float(TILE_SIZE_3DGS));
	float tiles_y = ceil(target.height / float(TILE_SIZE_3DGS));
	vec2 tileCoord = {
		pixelCoord.x / TILE_SIZE_3DGS,
		pixelCoord.y / TILE_SIZE_3DGS
	};

	if(tile_end.x < 0 || tile_start.x >= tiles_x) return;
	if(tile_end.y < 0 || tile_start.y >= tiles_y) return;

	tile_start.x = max(tile_start.x, 0);
	tile_end.x = min(tile_end.x, int(tiles_x) - 1);

	tile_start.y = max(tile_start.y, 0);
	tile_end.y = min(tile_end.y, int(tiles_y) - 1);

	color = applyColorCorrection(color, colorCorrection);

	uint32_t flags = model.flags[splatIndex];

	{ // HANDLE FLAGS

		bool isSelected           = (flags & FLAGS_SELECTED) != 0;
		bool isDeleted            = (flags & FLAGS_DELETED) != 0;
		bool isHighlighted        = (flags & FLAGS_HIGHLIGHTED) != 0;
		bool isHighlightedNeg     = (flags & FLAGS_HIGHLIGHTED_NEGATIVE) != 0;

		auto rect = args.rectselect;
		if(isDeleted && isHighlighted && args.brush.mode == BRUSHMODE::REMOVE_FLAGS){
			color.b = min(color.b * 2.0f + 0.4f, 1.0f);
			color.a = color.a / 4.0f;
		}else if(isDeleted){
			color.a = 0;
		}else if(isSelected && !isHighlightedNeg){
			color.r = min(color.r * 2.0f + 0.4f, 1.0f);
		}

		if(isHighlighted && (args.brush.mode == BRUSHMODE::ERASE)){
			color.r = min(color.r /  1.0f, 1.0f);
			color.g = min(color.g /  1.0f, 1.0f);
			color.b = min(color.b /  1.0f, 1.0f);
			color.a = min(color.a / 10.0f, 1.0f);
		}else if(isHighlighted && (args.brush.mode == BRUSHMODE::SELECT || rect.active)){
			color.r = min(color.r * 2.0f + 0.4f, 1.0f);
			color.g = min(color.g * 0.5f + 0.1f, 1.0f);
			color.b = min(color.b * 0.5f + 0.1f, 1.0f);
		}else if(isHighlighted && args.brush.mode == BRUSHMODE::REMOVE_FLAGS){
			color.r = min(color.r * 0.5f + 0.1f, 1.0f);
			color.g = min(color.g * 0.5f + 0.1f, 1.0f);
			color.b = min(color.b * 2.0f + 0.4f, 1.0f);
		}
	}

	if(!model.writeDepth){
		flags = flags | FLAGS_DISABLE_DEPTHWRITE;
	}

	if((flags & FLAGS_DELETED) != 0) return;

	// TODO: check if its more efficient to run a separate kernel that 
	// initializes <numVisibleSplats> elements with ordering[index] = index;
	uint32_t visibleSplatID = atomicAdd(visibleSplatCounter, 1);
	ordering[visibleSplatID] = visibleSplatID;

	StageData stuff;
	stuff.imgPos_encoded       = pixelCoord_encoded;
	stuff.flags                = flags;
	stuff.color                = to32BitColor(color);
	
	encode_stagedata(stuff, _basisVector1, _basisVector2, depth);

	staging_depth[visibleSplatID] = depth;
	staging_data[visibleSplatID] = stuff;

	// Count tile fragments that each splat produces
	uint32_t tileFrags = 0;

	forEachTouchedTile(pixelCoord, basisVector1, basisVector2, target, [&](uint32_t tile_x, uint32_t tile_y){
		tileFrags++;
	});

	numTilefragments_splatwise[visibleSplatID] = tileFrags;
	atomicAdd(numTilefragments, tileFrags);
}

inline constexpr int K = 16;
#define USE_FAST_CONFIG
#ifdef USE_FAST_CONFIG
// fast config
inline constexpr float MIN_ALPHA_THRESHOLD_RCP = 20.0f;
inline constexpr float MAX_CUTTOFF_SQ = 5.99146454711f; // logf(MIN_ALPHA_THRESHOLD_RCP * MIN_ALPHA_THRESHOLD_RCP)
#else
// reference config
inline constexpr float MIN_ALPHA_THRESHOLD_RCP = 255.0f;
inline constexpr float MAX_CUTTOFF_SQ = 11.0825270903f; // logf(MIN_ALPHA_THRESHOLD_RCP * MIN_ALPHA_THRESHOLD_RCP)
#endif
inline constexpr float MIN_ALPHA_THRESHOLD = 1.0f / MIN_ALPHA_THRESHOLD_RCP;
inline constexpr float MIN_ALPHA_THRESHOLD_CORE = (MIN_ALPHA_THRESHOLD_RCP >= 20.0f) ? 1.0f / 20.0f : MIN_ALPHA_THRESHOLD;
inline constexpr float MAX_FRAGMENT_ALPHA = 1.0f; // 3dgs uses 0.99f
inline constexpr float TRANSMITTANCE_THRESOLD = 1e-4f;

extern "C" __global__
void kernel_stageSplats_perspectivecorrect(
	CommonLaunchArgs args,
	RenderTarget target,
	ColorCorrection colorCorrection,
	GaussianData model,
	// out
	uint32_t* visibleSplatCounter,
	uint32_t* numTilefragments,
	uint32_t* numTilefragments_splatwise,
	float* staging_depth,
	glm::i16vec4* staging_bounds,
	StageData_perspectivecorrect* staging_data,
	uint32_t* ordering
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();
	int splatIndex = grid.thread_rank();
	constexpr int BLOCK_SIZE = TILE_SIZE_PERSPCORRECT * TILE_SIZE_PERSPCORRECT;

	if(splatIndex >= model.count) return;

	mat4 world = model.transform;
	mat4 view = target.view;
	mat4 worldView = view * world;

	vec4 splatPos = vec4(model.position[splatIndex], 1.0f);
	vec4 M3 = -glm::transpose(worldView)[2]; // times -1 because camera looks in negative z direction

	float z_view = dot(M3, splatPos);
	if (z_view < 0.2f || z_view > 1000.0f) return;

	vec4 color = model.color[splatIndex].normalized();
	if (color.a < MIN_ALPHA_THRESHOLD) return;
	vec4 quat = model.quaternion[splatIndex];
	mat3 rotation = glm::mat3_cast(glm::quat(quat.x, quat.y, quat.z, quat.w));

	mat3 scale = mat3(0.0f);
	scale[0][0] = model.scale[splatIndex].x;
	scale[1][1] = model.scale[splatIndex].y;
	scale[2][2] = model.scale[splatIndex].z;

	mat4 VP = target.VP;
	mat4 VPM = VP * worldView;
	mat3 RS = rotation * scale;
	mat4 T = mat4(RS);
	T[3] = splatPos;
	mat4 VPMT = glm::transpose(VPM * T);

	// tight cutoff for the used opacity threshold
	float rho_cutoff = 2.0f * logf(color.a * MIN_ALPHA_THRESHOLD_RCP);
	vec4 d = vec4(rho_cutoff, rho_cutoff, rho_cutoff, -1.0f);
	vec4 VPMT4 = VPMT[3];
	float s = dot(d, VPMT4 * VPMT4);
	if (s == 0.0f) return;
	vec4 f = (1.0f / s) * d;
	// start with z-extent in screen-space for exact near/far plane culling
	vec4 VPMT3 = VPMT[2];
	float center_z = dot(f, VPMT3 * VPMT4);
	float extent_z = sqrtf(fmaxf(center_z * center_z - dot(f, VPMT3 * VPMT3), 0.0f));
	float z_min = center_z - extent_z;
	float z_max = center_z + extent_z;
	if (z_min < -1.0f || z_max > 1.0f) return;
	// now x/y-extent of the screen-space bounding box
	vec4 VPMT1 = VPMT[0];
	vec4 VPMT2 = VPMT[1];
	vec2 center = vec2(dot(f, VPMT1 * VPMT4), dot(f, VPMT2 * VPMT4));
	vec2 extent = vec2(
		sqrtf(fmaxf(center.x * center.x - dot(f, VPMT1 * VPMT1), 0.0f)),
		sqrtf(fmaxf(center.y * center.y - dot(f, VPMT2 * VPMT2), 0.0f))
	);

	// if(args.uniforms.cullSmallSplats){
	// 	if(extent.x < 0.5f || extent.y < 0.5f) return;
	// }

	// compute screen-space bounding box in tile coordinates (+0.5 to account for half-pixel shift in V)
	int tiles_x = (target.width + TILE_SIZE_PERSPCORRECT - 1) / TILE_SIZE_PERSPCORRECT;
	int tiles_y = (target.height + TILE_SIZE_PERSPCORRECT - 1) / TILE_SIZE_PERSPCORRECT;
	ivec4 screen_bounds = vec4(
		min(tiles_x, max(0, __float2int_rd((center.x - extent.x + 0.5f) / TILE_SIZE_PERSPCORRECT))), // x_min (inclusive)
		min(tiles_y, max(0, __float2int_rd((center.y - extent.y + 0.5f) / TILE_SIZE_PERSPCORRECT))), // y_min (inclusive)
		min(tiles_x, max(0, __float2int_ru((center.x + extent.x + 0.5f) / TILE_SIZE_PERSPCORRECT))), // x_max (exclusive)
		min(tiles_y, max(0, __float2int_ru((center.y + extent.y + 0.5f) / TILE_SIZE_PERSPCORRECT))) // y_max (exclusive)
	);

	// compute number of potentially influenced tiles
	int n_touched_tiles = (screen_bounds.z - screen_bounds.x) * (screen_bounds.w - screen_bounds.y);
	if (n_touched_tiles == 0) return;

	color = applyColorCorrection(color, colorCorrection);

	uint32_t flags = model.flags[splatIndex];

	{ // HANDLE FLAGS

		bool isSelected           = (flags & FLAGS_SELECTED) != 0;
		bool isDeleted            = (flags & FLAGS_DELETED) != 0;
		bool isHighlighted        = (flags & FLAGS_HIGHLIGHTED) != 0;
		bool isHighlightedNeg     = (flags & FLAGS_HIGHLIGHTED_NEGATIVE) != 0;

		auto rect = args.rectselect;
		if(isDeleted && isHighlighted && args.brush.mode == BRUSHMODE::REMOVE_FLAGS){
			color.b = min(color.b * 2.0f + 0.4f, 1.0f);
			color.a = color.a / 4.0f;
		}else if(isDeleted){
			color.a = 0;
		}else if(isSelected && !isHighlightedNeg){
			color.r = min(color.r * 2.0f + 0.4f, 1.0f);
		}

		if(isHighlighted && (args.brush.mode == BRUSHMODE::ERASE)){
			color.r = min(color.r /  1.0f, 1.0f);
			color.g = min(color.g /  1.0f, 1.0f);
			color.b = min(color.b /  1.0f, 1.0f);
			color.a = min(color.a / 10.0f, 1.0f);
		}else if(isHighlighted && (args.brush.mode == BRUSHMODE::SELECT || rect.active)){
			color.r = min(color.r * 2.0f + 0.4f, 1.0f);
			color.g = min(color.g * 0.5f + 0.1f, 1.0f);
			color.b = min(color.b * 0.5f + 0.1f, 1.0f);
		}else if(isHighlighted && args.brush.mode == BRUSHMODE::REMOVE_FLAGS){
			color.r = min(color.r * 0.5f + 0.1f, 1.0f);
			color.g = min(color.g * 0.5f + 0.1f, 1.0f);
			color.b = min(color.b * 2.0f + 0.4f, 1.0f);
		}
	}

	if(!model.writeDepth){
		flags = flags | FLAGS_DISABLE_DEPTHWRITE;
	}

	if((flags & FLAGS_DELETED) != 0) return;

	uint32_t visibleSplatID = atomicAdd(visibleSplatCounter, 1);
	ordering[visibleSplatID] = visibleSplatID;

	StageData_perspectivecorrect stuff;
	stuff.VPMT1 = VPMT1;
	stuff.VPMT2 = VPMT2;
	stuff.VPMT4 = VPMT4;
	stuff.MT3 = vec4(dot(M3, T[0]), dot(M3, T[1]), dot(M3, T[2]), z_view);
	stuff.color = to32BitColor(color);
	stuff.flags = flags;

	staging_depth[visibleSplatID] = z_view;
	staging_bounds[visibleSplatID] = glm::i16vec4(screen_bounds);
	staging_data[visibleSplatID] = stuff;

	numTilefragments_splatwise[visibleSplatID] = n_touched_tiles;
	atomicAdd(numTilefragments, n_touched_tiles);

}

extern "C" __global__
void kernel_applyOrdering_u32(uint32_t* unsorted, uint32_t* sorted, uint32_t* ordering, uint32_t count){
	uint32_t index = cg::this_grid().thread_rank();

	if(index >= count) return;

	uint32_t order = ordering[index];
	sorted[index] = unsorted[order];
}

extern "C" __global__
void kernel_applyOrdering_xxx(uint8_t* unsorted, uint8_t* sorted, uint32_t* ordering, uint64_t stride, uint32_t count){
	uint64_t index = cg::this_grid().thread_rank();

	if(index >= count) return;

	uint64_t sourceOffset = uint64_t(ordering[index]) * stride;
	uint64_t targetOffset = index * stride;

	memcpy(sorted + targetOffset, unsorted + sourceOffset, stride);
}


extern "C" __global__
void kernel_applyOrdering_stagedata(StageData* unsorted, StageData* sorted, uint32_t* ordering, uint32_t count){
	uint32_t index = cg::this_grid().thread_rank();

	if(index >= count) return;

	sorted[index] = unsorted[ordering[index]];
}

extern "C" __global__
void kernel_createTilefragmentArray(
	// input
	CommonLaunchArgs args,
	RenderTarget target,
	uint32_t* ordering, 
	uint32_t numStagedSplats,
	StageData* stageDataArray,
	// uint32_t* dbg_numTilefragments_ordered,
	uint32_t* prefixsum,
	uint32_t tileFragmentsCounter,
	// output
	uint32_t* tileIDs,
	uint32_t* splatIDs
){
	
	// index of unsorted, staged splats
	uint32_t index = cg::this_grid().thread_rank();

	if(index >= numStagedSplats) return;

	// index of depth-sorted splats
	uint32_t order = ordering[index];

	// load stagedata of splats in depth-sorted order
	StageData stageData = stageDataArray[order];

	vec2 basisVector1, basisVector2;
	float depth;
	decode_stagedata(stageData, basisVector1, basisVector2, depth);

	vec2 pixelCoord = vec2(stageData.imgPos_encoded) / 10.0f;

	float quadHalfWidth  = sqrt(basisVector1.x * basisVector1.x + basisVector2.x * basisVector2.x);
	float quadHalfHeight = sqrt(basisVector1.y * basisVector1.y + basisVector2.y * basisVector2.y);

	float tiles_x = ceil(target.width / float(TILE_SIZE_3DGS));
	float tiles_y = ceil(target.height / float(TILE_SIZE_3DGS));

	ivec2 tile_start = {
		(pixelCoord.x - quadHalfWidth) / TILE_SIZE_3DGS,
		(pixelCoord.y - quadHalfHeight) / TILE_SIZE_3DGS};
	ivec2 tile_end = {
		(pixelCoord.x + quadHalfWidth) / TILE_SIZE_3DGS,
		(pixelCoord.y + quadHalfHeight) / TILE_SIZE_3DGS};

	tile_start.x = max(tile_start.x, 0);
	tile_end.x = min(tile_end.x, int(tiles_x) - 1);

	tile_start.y = max(tile_start.y, 0);
	tile_end.y = min(tile_end.y, int(tiles_y) - 1);

	int ltiles_x = (tile_end.x - tile_start.x) + 1;
	int ltiles_y = (tile_end.y - tile_start.y) + 1;
	int numTiles = ltiles_x * ltiles_y;

	uint32_t fragmentOffset = prefixsum[index];

	forEachTouchedTile(pixelCoord, basisVector1, basisVector2, target, [&](uint32_t tile_x, uint32_t tile_y){
		uint32_t tileID = tile_x + tile_y * tiles_x;

		tileIDs[fragmentOffset] = tileID;
		splatIDs[fragmentOffset] = order;

		fragmentOffset++;
	});
	
}

extern "C" __global__
void kernel_createTilefragmentArray_perspectivecorrect(
	// input
	CommonLaunchArgs args,
	RenderTarget target,
	uint32_t* ordering, 
	uint32_t numStagedSplats,
	glm::i16vec4* staging_bounds,
	uint32_t* prefixsum,
	uint32_t tileFragmentsCounter,
	// output
	uint32_t* tileIDs,
	uint32_t* splatIDs
){
	// index of unsorted, staged splats
	uint32_t index = cg::this_grid().thread_rank();

	if(index >= numStagedSplats) return;

	// index of depth-sorted splats
	uint32_t order = ordering[index];

	// load bounds of splats in depth-sorted order
	glm::i16vec4 bounds = staging_bounds[order];

	uint32_t fragmentOffset = prefixsum[index];

	uint32_t tiles_x = (target.width + TILE_SIZE_PERSPCORRECT - 1) / TILE_SIZE_PERSPCORRECT;

	for (uint32_t tile_x = bounds.x; tile_x < bounds.z; tile_x++)
	for (uint32_t tile_y = bounds.y; tile_y < bounds.w; tile_y++)
	{
		uint32_t tileID = tile_y * tiles_x + tile_x;

		tileIDs[fragmentOffset] = tileID;
		splatIDs[fragmentOffset] = order;

		fragmentOffset++;
	}
}


extern "C" __global__
void kernel_prefilter_tiled_stagedata(
	Tile* tiles, 
	uint32_t* indices, 
	StageData* stagedatas
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	int tileID = grid.block_rank();

	Tile tile = tiles[tileID];

	int numPointsInTile = clamp(tile.lastIndex - tile.firstIndex + 1, 0u, 1'000'000u);

	if(numPointsInTile < 10'000) return;

	int iterations = ((numPointsInTile + block.num_threads() - 1) / block.num_threads());

	for(int iteration = 0; iteration < iterations; iteration++){
		
		int index = block.num_threads() * iteration + block.thread_rank();
		
		if(index < numPointsInTile){
			int splatIndex = indices[tile.firstIndex + index];
			StageData stageData = stagedatas[splatIndex];

			uint32_t C = stageData.color;

			vec2 basisvector1, basisvector2;
			float depth;
			decode_stagedata(stageData, basisvector1, basisvector2, depth);

			float opacity = ((C >> 24) & 0xff) / 255.0f;

			// float lx = fmodf(data.imgpos.x, 16.0f);
			// float ly = fmodf(data.imgpos.y, 16.0f);
			// int lindex = int(lx) + 16 * int(ly);
			// bool hasLowOpacity = oapcity < 0.5f;
			// bool isSmall = length(basisvector1) < 10.0f && length(basisvector2) < 10.0f;
			// bool isRejected = isSmall && hasLowOpacity && sh_remainingTranslucency[lindex] <= 0.1f;
			// bool isAccepted = !isRejected;

			bool isAccepted = opacity > 0.002f;

			if(!isAccepted){
				stageData.color = stageData.color | 0xff0000ff;
				stagedatas[splatIndex] = stageData;
			}


		}
	}

	
}



// this method uses the same approach as INRIA / Kerbl & Kopanas et al.
// rasterizer_impl method identifyTileRanges()
// https://github.com/graphdeco-inria/gaussian-splatting
extern "C" __global__
void kernel_computeTiles_method1(
	CommonLaunchArgs args, RenderTarget target, 
	uint32_t* tileIDs, uint32_t numFragments,
	int tile_size,
	// output
	Tile* tiles
){
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if(index >= numFragments) return;

	int tiles_x = (target.width + tile_size - 1) / tile_size;
	int tiles_y = (target.height + tile_size - 1) / tile_size;
	uint32_t numTiles = tiles_x * tiles_y;


	uint32_t tileID = tileIDs[index];

	if(tileID >= numTiles) return;

	if(index == 0){
		tiles[tileID].firstIndex = 0;
	}else{
		uint32_t prevTileID = tileIDs[index - 1];

		if(tileID != prevTileID){
			tiles[prevTileID].lastIndex = index - 1;
			tiles[tileID].firstIndex = index;
		}
	}

	if(index == numFragments - 1){
		tiles[tileID].lastIndex = index;
	}
}

extern "C" __global__
void kernel_clearFramebuffer(CommonLaunchArgs args, RenderTarget target){

	int pixelID = cg::this_grid().thread_rank();
	int numPixels = target.width * target.height;

	if(pixelID >= numPixels) return;

	target.framebuffer[pixelID] = DEFAULT_PIXEL;
}

extern "C" __global__
void kernel_clearDepthbuffer(CommonLaunchArgs args, RenderTarget target){

	int pixelID = cg::this_grid().thread_rank();
	int numPixels = target.width * target.height;

	if(pixelID >= numPixels) return;

	Pixel* framebuffer = (Pixel*)target.framebuffer;

	Pixel pixel = framebuffer[pixelID];
	pixel.depth = Infinity;

	framebuffer[pixelID] = pixel;

	if(target.indexbuffer){
		target.indexbuffer[pixelID] = DEFAULT_PIXEL;
	}
}

extern "C" __global__
void kernel_toOpenGL(
	CommonLaunchArgs args,
	RenderTarget target,
	hipSurfaceObject_t gl_colorbuffer
){
	auto grid = cg::this_grid();

	int width = target.width;
	int height = target.height;

	int pixelID = grid.thread_rank();
	int x = pixelID % width;
	int y = pixelID / width;

	if(pixelID >= width * height) return;

	uint64_t pixel = target.framebuffer[pixelID];
	uint32_t color = pixel & 0xffffffff;

	{ // show debug tile
		int tiles_x = (target.width + 16 - 1) / 16;
		int tile_x = x / 16;
		int tile_y = y / 16;
		int tileID = tile_x + tile_y * tiles_x;
		// if(isDebugTile(tile_x, tile_y))
		// if(x % 16 == 0 || x % 16 == 15 || y % 16 == 0 || y % 16 == 15)
		// {
		// 	color = 0xff0000ff;
		// }

		if(args.uniforms.inset.show){
			if(x == dbgtile_x * 16 + 16){
				color = 0xffff00ff;
			}
			if(x == dbgtile_x * 16 - 1){
				color = 0xffff00ff;
			}
			if(y == dbgtile_y * 16 - 1){
				color = 0xffff00ff;
			}
			if(y == dbgtile_y * 16 + 16){
				color = 0xffff00ff;
			}
		}
	}

	// color = 0xff0000ff;
	surf2Dwrite(color, gl_colorbuffer, x * 4, y);
}

extern "C" __global__
void kernel_blit_opengl(
	CommonLaunchArgs args,
	RenderTarget source,
	hipSurfaceObject_t gl_colorbuffer,
	Rectangle target
){
	auto grid = cg::this_grid();

	int pixelID = grid.thread_rank();
	int target_relative_x = pixelID % int(target.width);
	int target_relative_y = pixelID / int(target.width);

	if(pixelID >= target.width * target.height) return;

	int target_x = target.x + target_relative_x;
	int target_y = target.y + target_relative_y;

	float u = float(target_relative_x + 0.5f) / float(target.width);
	float v = float(target_relative_y + 0.5f) / float(target.height);

	int source_x = clamp(float(source.width) * u, 0.0f, source.width - 1.0f);
	int source_y = clamp(float(source.height) * v, 0.0f, source.height - 1.0f);
	int source_pixelID = source_x + source_y * source.width;

	uint64_t sourcePixel = source.framebuffer[source_pixelID];
	uint32_t color = sourcePixel & 0xffffffff;

	surf2Dwrite(color, gl_colorbuffer, target_x * 4, target_y);
}

// extern "C" __global__
// void kernel_drawPoints_depth(CommonLaunchArgs args, GaussianData splats, RenderTarget target, uint32_t* depthbuffer, uint32_t* accumlatebuffer){

// 	auto grid = cg::this_grid();

// 	int index = grid.thread_rank();

// 	if(index >= splats.count) return;
	
// 	vec3 pos = splats.position[index];
// 	uint32_t flags = splats.flags[index];
// 	vec4 color = splats.color[index];
// 	float opacity = color.a;

// 	if(opacity < 10.0f) return;

// 	mat4 transform = target.proj * target.view * splats.transform;
// 	vec4 ndc = transform * vec4(pos, 1.0f);
// 	vec2 imgCoords = vec2(
// 		(0.5f * (ndc.x / ndc.w) + 0.5f) * target.width,
// 		(0.5f * (ndc.y / ndc.w) + 0.5f) * target.height
// 	);

// 	if(ndc.w < 0.0f) return;

// 	float size = 0.0f;

// 	for(float dx = -size; dx <= size; dx += 1.0f)
// 	for(float dy = -size; dy <= size; dy += 1.0f)
// 	{
// 		vec2 coord = imgCoords + vec2{dx, dy};

// 		if(coord.x < 0.0f || coord.x >= target.width) continue;
// 		if(coord.y < 0.0f || coord.y >= target.height) continue;

// 		int pixelID = int(coord.x) + int(coord.y) * target.width;

// 		uint32_t udepth = __float_as_uint(ndc.w);
// 		uint32_t oldDepth = depthbuffer[pixelID];

// 		if(udepth < oldDepth){
// 			atomicMin(&depthbuffer[pixelID], udepth);
// 		}
// 	}
// }

// extern "C" __global__
// void kernel_drawPoints_accumulate(CommonLaunchArgs args, GaussianData splats, RenderTarget target, float* depthbuffer, uint32_t* accumlatebuffer){

// 	auto grid = cg::this_grid();

// 	int index = grid.thread_rank();

// 	if(index >= splats.count) return;
	
// 	vec3 pos = splats.position[index];
// 	uint32_t flags = splats.flags[index];
// 	vec4 color = splats.color[index];
// 	// uint8_t* rgba = (uint8_t*)&color;
// 	float opacity = color.a;

// 	if(opacity < 1.0f) return;

// 	mat4 transform = target.proj * target.view * splats.transform;
// 	vec4 ndc = transform * vec4(pos, 1.0f);
// 	vec2 imgCoords = vec2(
// 		(0.5f * (ndc.x / ndc.w) + 0.5f) * target.width,
// 		(0.5f * (ndc.y / ndc.w) + 0.5f) * target.height
// 	);

// 	if(ndc.w < 0.0f) return;

// 	float size = 0.0f;

// 	for(float dx = -size; dx <= size; dx += 1.0f)
// 	for(float dy = -size; dy <= size; dy += 1.0f)
// 	{
// 		vec2 coord = imgCoords + vec2{dx, dy};

// 		if(coord.x < 0.0f || coord.x >= target.width) continue;
// 		if(coord.y < 0.0f || coord.y >= target.height) continue;

// 		int pixelID = int(coord.x) + int(coord.y) * target.width;

// 		uint32_t udepth = __float_as_uint(ndc.w);
// 		float oldDepth = depthbuffer[pixelID] * 1.01f;
// 		uint32_t uOldDepth = __float_as_int(oldDepth);


// 		if(udepth < uOldDepth){
// 			atomicAdd(&accumlatebuffer[4 * pixelID + 0], uint32_t(opacity * color.r * 255.0f));
// 			atomicAdd(&accumlatebuffer[4 * pixelID + 1], uint32_t(opacity * color.g * 255.0f));
// 			atomicAdd(&accumlatebuffer[4 * pixelID + 2], uint32_t(opacity * color.b * 255.0f));
// 			atomicAdd(&accumlatebuffer[4 * pixelID + 3], uint32_t(opacity * 255.0f));
// 		}
// 	}
// }

// extern "C" __global__
// void kernel_drawPoints_resolve(
// 	CommonLaunchArgs args, GaussianData splats, RenderTarget target, float* depthbuffer, 
// 	uint32_t* accumlatebuffer){

// 	auto grid = cg::this_grid();

// 	int index = grid.thread_rank();

// 	if(index >= target.width * target.height) return;
	
// 	int px = index % target.width;
// 	int py = index / target.width;
// 	int pixelID = px + py * target.width;

// 	int R = accumlatebuffer[4 * pixelID + 0];
// 	int G = accumlatebuffer[4 * pixelID + 1];
// 	int B = accumlatebuffer[4 * pixelID + 2];
// 	int count = accumlatebuffer[4 * pixelID + 3];

// 	int r = R / count;
// 	int g = G / count;
// 	int b = B / count;

// 	uint32_t color;
// 	uint8_t* rgba = (uint8_t*)&color;
// 	rgba[0] = r;
// 	rgba[1] = g;
// 	rgba[2] = b;

// 	float depth = depthbuffer[pixelID];
// 	uint64_t udepth = __float_as_uint(depth);

// 	uint64_t pixel = (udepth << 32) | color;
	
// 	uint64_t oldPixel = target.framebuffer[pixelID];
// 	uint32_t oldDepth = oldPixel >> 32;
// 	uint32_t oldColor = oldPixel & 0xffffffff;

// 	if(udepth < oldDepth){
// 		target.framebuffer[pixelID] = pixel;
// 	}
// }

template<typename T>
__device__ void swap(
    T& a,
    T& b)
{
    T temp = a;
    a = b;
    b = temp;
}

extern "C" __global__
void kernel_render_gaussians_perspectivecorrect(
	CommonLaunchArgs args, RenderTarget target,
	Tile* tiles, uint32_t* indices, StageData_perspectivecorrect* stagedatas
){
	auto grid     = cg::this_grid();
	auto block    = cg::this_thread_block();
	constexpr int BLOCK_SIZE = TILE_SIZE_PERSPCORRECT * TILE_SIZE_PERSPCORRECT;

	dim3 group_index = block.group_index();
	dim3 thread_index = block.thread_index();
	uint32_t thread_rank = block.thread_rank();
	uint2 pixel_coords = make_uint2(group_index.x * TILE_SIZE_PERSPCORRECT + thread_index.x, group_index.y * TILE_SIZE_PERSPCORRECT + thread_index.y);
	int width     = target.width;
	int height    = target.height;
	int tiles_x   = (width + TILE_SIZE_PERSPCORRECT - 1) / TILE_SIZE_PERSPCORRECT;
	int tiles_y   = (height + TILE_SIZE_PERSPCORRECT - 1) / TILE_SIZE_PERSPCORRECT;

	int tileID = grid.block_rank();
	
	Tile tile = tiles[tileID];

	// tile coordinates
	int tile_x = tileID % tiles_x;
	int tile_y = tileID / tiles_x;

	// this thread's pixel coordinate within tile
	int tilePixelIndex = threadIdx.x;
	int tile_pixel_x = tilePixelIndex % TILE_SIZE_PERSPCORRECT;
	int tile_pixel_y = tilePixelIndex / TILE_SIZE_PERSPCORRECT;

	// This thread's pixel coordinates within framebuffer
	int pixel_x = tile_x * TILE_SIZE_PERSPCORRECT + tile_pixel_x;
	int pixel_y = tile_y * TILE_SIZE_PERSPCORRECT + tile_pixel_y;
	float fpixel_x = __int2float_rn(pixel_x);
	float fpixel_y = __int2float_rn(pixel_y);
	int pixelID = pixel_x + pixel_y * width;

	bool inside = pixel_x < width && pixel_y < height;
	// setup shared memory
	__shared__ vec4 collected_VPMT1[BLOCK_SIZE], collected_VPMT2[BLOCK_SIZE], collected_VPMT4[BLOCK_SIZE], collected_MT3[BLOCK_SIZE];
	__shared__ vec3 collected_rgb[BLOCK_SIZE];
	__shared__ float collected_opacity[BLOCK_SIZE];
	__shared__ uint32_t collected_flags[BLOCK_SIZE];
	// initialize local storage
	float transmittance_tail = 1.0f;
	vec4 rgba_premultiplied_tail = vec4(0.0f);
	__half2 rgbas_premultiplied_core_rg[K];
	__half2 rgbas_premultiplied_core_ba[K];
	float depths_core[K];
	bool validForDepth[K];
	#pragma unroll
	for (int i = 0; i < K; ++i) {
		rgbas_premultiplied_core_rg[i]= {0};
		rgbas_premultiplied_core_ba[i]= {0};
		depths_core[i] = __FLT_MAX__;
		validForDepth[i] = true;
	}
	float oldDepth = inside ? __int_as_float(target.framebuffer[pixelID] >> 32) : Infinity;
	// collaborative loading and processing
	uint2 tile_range = make_uint2(tile.firstIndex, tile.lastIndex + 1); // TODO: lastIndex or lastIndex + 1?
	for (int n_points_remaining = tile_range.y - tile_range.x, current_fetch_idx = tile_range.x + thread_rank; n_points_remaining > 0; n_points_remaining -= BLOCK_SIZE, current_fetch_idx += BLOCK_SIZE) {
		block.sync();
		if (current_fetch_idx < tile_range.y) {
			int splatIndex = indices[current_fetch_idx];
			StageData_perspectivecorrect stageData = stagedatas[splatIndex];
			uint32_t C = stageData.color;
			vec4 rgba = vec4{
				(C >>  0) & 0xff,
				(C >>  8) & 0xff,
				(C >> 16) & 0xff,
				(C >> 24) & 0xff,
			 } / 255.0f;
			collected_VPMT1[thread_rank] = stageData.VPMT1;
			collected_VPMT2[thread_rank] = stageData.VPMT2;
			collected_VPMT4[thread_rank] = stageData.VPMT4;
			collected_MT3[thread_rank] = stageData.MT3;
			collected_rgb[thread_rank] = vec3(rgba);
			collected_opacity[thread_rank] = rgba.w;
			collected_flags[thread_rank] = stageData.flags;
		}
		block.sync();
		if (inside) {
			int current_batch_size = min(BLOCK_SIZE, n_points_remaining);
			for (int j = 0; j < current_batch_size; ++j) {
				vec4 VPMT1 = collected_VPMT1[j];
				vec4 VPMT2 = collected_VPMT2[j];
				vec4 VPMT4 = collected_VPMT4[j];
				vec4 plane_x_diag = VPMT1 - VPMT4 * fpixel_x;
				vec4 plane_y_diag = VPMT2 - VPMT4 * fpixel_y;
				vec3 plane_x_diag_normal = vec3(plane_x_diag);
				vec3 plane_y_diag_normal = vec3(plane_y_diag);
				vec3 m = plane_x_diag.w * plane_y_diag_normal - plane_x_diag_normal * plane_y_diag.w;
				vec3 d = cross(plane_x_diag_normal, plane_y_diag_normal);
				float numerator_rho2 = dot(m, m);
				float denominator = dot(d, d);
				if (numerator_rho2 > MAX_CUTTOFF_SQ * denominator) continue; // considering opacity requires log/sqrt -> slower
				float denominator_rcp = 1.0f / denominator;
				float G = expf(-0.5f * numerator_rho2 * denominator_rcp);
				float opacity = collected_opacity[j];
				float alpha = fminf(opacity * G, MAX_FRAGMENT_ALPHA);
				if (alpha < MIN_ALPHA_THRESHOLD) continue;
				if (args.uniforms.showRing && alpha < 0.1f){
					alpha += 0.9f;
				}

				vec3 eval_point_diag = cross(d, m) * denominator_rcp;
				vec4 MT3 = collected_MT3[j];
				float depth = dot(vec3(MT3), eval_point_diag) + MT3.w;
				if (depth >= oldDepth) continue;

				bool depth_valid = (collected_flags[j] & FLAGS_DISABLE_DEPTHWRITE) == 0;
				
				vec3 rgb = collected_rgb[j];
				__half2 rgba_premultiplied_rg = __float22half2_rn(make_float2(rgb.x * alpha, rgb.y * alpha));
				__half2 rgba_premultiplied_ba = __float22half2_rn(make_float2(rgb.z * alpha, alpha));
				if (depth < depths_core[K - 1] && alpha >= MIN_ALPHA_THRESHOLD_CORE) {
					#pragma unroll
					for (int core_idx = 0; core_idx < K; ++core_idx) {
						if (depth < depths_core[core_idx]) {
							swap(depth, depths_core[core_idx]);
							swap(rgba_premultiplied_rg, rgbas_premultiplied_core_rg[core_idx]);
							swap(rgba_premultiplied_ba, rgbas_premultiplied_core_ba[core_idx]);
							swap(depth_valid, validForDepth[core_idx]);
						}
					}
				}
				rgba_premultiplied_tail += vec4(__half2float(rgba_premultiplied_rg.x), __half2float(rgba_premultiplied_rg.y), __half2float(rgba_premultiplied_ba.x), __half2float(rgba_premultiplied_ba.y));
				transmittance_tail *= 1.0f - __half2float(rgba_premultiplied_ba.y);
			}
		}
	}
	if (inside) {
		// blend core
		vec3 rgb_pixel = vec3(0.0f);
		float depth_pixel = __FLT_MAX__;
		float transmittance_core = 1.0f;
		bool done = false;
		#pragma unroll
		for (int core_idx = 0; core_idx < K && !done; ++core_idx) {
			float2 rgba_premultiplied_rg = __half22float2(rgbas_premultiplied_core_rg[core_idx]);
			float2 rgba_premultiplied_ba = __half22float2(rgbas_premultiplied_core_ba[core_idx]);
			vec3 rgb_premultiplied = vec3(rgba_premultiplied_rg.x, rgba_premultiplied_rg.y, rgba_premultiplied_ba.x);

			float depth = depths_core[core_idx];
			depth_pixel = (transmittance_core > 0.5f && depth < __FLT_MAX__ && validForDepth[core_idx]) ? depth : depth_pixel;

			rgb_pixel += transmittance_core * rgb_premultiplied;
			transmittance_core *= 1.0f - rgba_premultiplied_ba.y;
			if (transmittance_core < TRANSMITTANCE_THRESOLD) done = true;
		}

		float transmittance_pixel = transmittance_core;
		// blend tail
		if (!done && rgba_premultiplied_tail.w >= MIN_ALPHA_THRESHOLD) {
			float weight_tail = transmittance_core * (1.0f - transmittance_tail);
			rgb_pixel += weight_tail * (1.0f / rgba_premultiplied_tail.w) * vec3(rgba_premultiplied_tail);
			transmittance_pixel *= transmittance_tail;
		}

		uint32_t color = 0;
		uint8_t* rgba = (uint8_t*)&color;
		if(args.uniforms.rendermode == RENDERMODE_COLOR){

			uint32_t C = target.framebuffer[pixelID] & 0xffffffff;
			vec4 oldPixel = vec4{
				(C >>  0) & 0xff,
				(C >>  8) & 0xff,
				(C >> 16) & 0xff,
				(C >> 24) & 0xff,
			};

			rgba[0] = clamp(rgb_pixel.r * 255.0f + transmittance_pixel * oldPixel.r, 0.0f, 255.0f);
			rgba[1] = clamp(rgb_pixel.g * 255.0f + transmittance_pixel * oldPixel.g, 0.0f, 255.0f);
			rgba[2] = clamp(rgb_pixel.b * 255.0f + transmittance_pixel * oldPixel.b, 0.0f, 255.0f);
			rgba[3] = 255;

			uint32_t udepth = __float_as_uint(depth_pixel);
			uint64_t pixelValue = (uint64_t(udepth) << 32) | color;

			target.framebuffer[pixelID] = pixelValue;
		} else if(args.uniforms.rendermode == RENDERMODE_DEPTH){

			vec3 SPECTRAL[11] = {
				vec3{158,1,66},
				vec3{213,62,79},
				vec3{244,109,67},
				vec3{253,174,97},
				vec3{254,224,139},
				vec3{255,255,191},
				vec3{230,245,152},
				vec3{171,221,164},
				vec3{102,194,165},
				vec3{50,136,189},
				vec3{94,79,162},
			};

			float w = depth_pixel / 3.0f;
			float u = w - floor(w);

			int i0 = w;

			vec3 C0 = SPECTRAL[clamp((i0 + 0), 0, 10) % 11];
			vec3 C1 = SPECTRAL[clamp((i0 + 1), 0, 10) % 11];

			vec3 C = (1.0f - u) * C0 + u * C1;

			rgba[0] = C.x;
			rgba[1] = C.y;
			rgba[2] = C.z;

			uint32_t udepth = __float_as_uint(depth_pixel);

			target.framebuffer[pixelID] = (uint64_t(udepth) << 32) | color;
		}
	}

}


#include "./render/render_gaussians.cuh"
// #include "./render/render_gaussians_subsets.cuh"
#include "./render/render_gaussians_solid.cuh"
// #include "./render/render_gaussians_fetchfilter.cuh"
#include "./render/render_heatmap.cuh"